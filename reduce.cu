
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void SerialKernel(const float* input, float* output, size_t n) {
  float sum = 0.0f;  
  for (size_t i = 0; i < n; ++i) {  
    sum += input[i];  
  }  
  *output = sum;  
}

void ReduceBySerial(const float* input, float* output, size_t n) {
  SerialKernel<<<1, 1>>>(input, output, n);
}

__global__ void AtomicKernel(const float* input, float* output, size_t n) {
  int32_t gtid = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
  atomicAdd(output,input[gtid]);
}

void ReduceByAtomic(const float* input, float* output, size_t n) {
    AtomicKernel<<<n/1024, 1024>>>(input, output, n);
}

__global__ void TwoPassSimpleKernel(const float* input, float* part_sum,
                                    size_t n) {
  // n is divided to gridDim.x part
  // this block process input[blk_begin:blk_end]
  // store result to part_sum[blockIdx.x]
  size_t blk_begin = n / gridDim.x * blockIdx.x;
  size_t blk_end = n / gridDim.x * (blockIdx.x + 1);
  // after follow step, this block process input[0:n], store result to part_sum
  n = blk_end - blk_begin;
  input += blk_begin;
  part_sum += blockIdx.x;
  // n is divided to blockDim.x part
  // this thread process input[thr_begin:thr_end]
  size_t thr_begin = n / blockDim.x * threadIdx.x;
  size_t thr_end = n / blockDim.x * (threadIdx.x + 1);
  float thr_sum = 0.0f;
  for (size_t i = thr_begin; i < thr_end; ++i) {
    thr_sum += input[i];
  }
  // store thr_sum to shared memory
  extern __shared__ float shm[];
  shm[threadIdx.x] = thr_sum;
  __syncthreads();
  // reduce shm to part_sum
  if (threadIdx.x == 0) {
    float sum = 0.0f;
    for (size_t i = 0; i < blockDim.x; ++i) {
      sum += shm[i];
    }
    *part_sum = sum;
  }
}

void ReduceByTwoPass(const float* input, float* output, size_t n) {
  const int32_t thread_num_per_block = 1024;  // tuned
  const int32_t block_num = 1024;             // tuned
  float *part = NULL;
  hipMalloc((void**)&part,(1024*sizeof(float)));
  // the first pass reduce input[0:n] to part[0:block_num]
  // part_sum[i] stands for the result of i-th block
  size_t shm_size = thread_num_per_block * sizeof(float);  // float per thread
  TwoPassSimpleKernel<<<block_num, thread_num_per_block, shm_size>>>(input,part, n);
  // the second pass reduce part[0:block_num] to output
  TwoPassSimpleKernel<<<1, thread_num_per_block, shm_size>>>(part, output, block_num);
}



// warp divergence
__global__ void TwoPassInterleavedKernel(const float* input, float* part_sum,
                                         size_t n) {
  int32_t gtid = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
  int32_t total_thread_num = gridDim.x * blockDim.x;
  // reduce
  //   input[gtid + total_thread_num * 0]
  //   input[gtid + total_thread_num * 1]
  //   input[gtid + total_thread_num * 2]
  //   input[gtid + total_thread_num * ...]
  float sum = 0.0f;
  for (int32_t i = gtid; i < n; i += total_thread_num) {
    sum += input[i];
  }
  // store sum to shared memory
  extern __shared__ float shm[];
  shm[threadIdx.x] = sum;
  __syncthreads();
  // reduce shm to part_sum
  if (threadIdx.x == 0) {
    float sum = 0.0f;
    for (size_t i = 0; i < blockDim.x; ++i) {
      sum += shm[i];
    }
    part_sum[blockIdx.x] = sum;
  }
}


void ReduceByTwoPassInterleaved(const float* input, float* output, size_t n) {
  const int32_t thread_num_per_block = 1024;  // tuned
  const int32_t block_num = 1024;             // tuned
  float *part = NULL;
  hipMalloc((void**)&part,(1024*sizeof(float)));
  // the first pass reduce input[0:n] to part[0:block_num]
  // part_sum[i] stands for the result of i-th block
  size_t shm_size = thread_num_per_block * sizeof(float);  // float per thread
  TwoPassInterleavedKernel<<<block_num, thread_num_per_block, shm_size>>>(input,part, n);
  // the second pass reduce part[0:block_num] to output
  TwoPassInterleavedKernel<<<1, thread_num_per_block, shm_size>>>(part, output, block_num);
}

__global__ void baselineKernel(const float* input, float* part_sum,
                                         size_t n) {
  int32_t gtid = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
  int32_t total_thread_num = gridDim.x * blockDim.x;
  float sum = 0.0f;
  for (int32_t i = gtid; i < n; i += total_thread_num) {
    sum += input[i];
  }
  // store sum to shared memory
  extern __shared__ float shm[];
  shm[threadIdx.x] = sum;
  __syncthreads();
  // reduce shm to part_sum
  for(unsigned int s=1; s<blockDim.x; s*=2){
    if(threadIdx.x%(2*s) == 0){
        shm[threadIdx.x]+=shm[threadIdx.x+s];
    }
    __syncthreads();
  }
    
 if (threadIdx.x == 0) part_sum[blockIdx.x] = shm[0];
}

void ReduceByBaseline(const float* input, float* output, size_t n) {
  const int32_t thread_num_per_block = 1024;  // tuned
  const int32_t block_num = 1024;             // tuned
  float *part = NULL;
  hipMalloc((void**)&part,(1024*sizeof(float)));
  // the first pass reduce input[0:n] to part[0:block_num]
  // part_sum[i] stands for the result of i-th block
  size_t shm_size = thread_num_per_block * sizeof(float);  // float per thread
  baselineKernel<<<block_num, thread_num_per_block, shm_size>>>(input,part, n);
  // the second pass reduce part[0:block_num] to output
  baselineKernel<<<1, thread_num_per_block, shm_size>>>(part, output, block_num);
}

__global__ void warpBranchKernel(const float* input, float* part_sum,
                                         size_t n) {
  int32_t gtid = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
  int32_t total_thread_num = gridDim.x * blockDim.x;
  float sum = 0.0f;
  for (int32_t i = gtid; i < n; i += total_thread_num) {
    sum += input[i];
  }
  // store sum to shared memory
  extern __shared__ float shm[];
  shm[threadIdx.x] = sum;
  __syncthreads();
  // reduce shm to part_sum
  for(unsigned int s=1; s<blockDim.x; s*=2){
    int index = 2*s*threadIdx.x;
    if(index < blockDim.x){
        shm[index]+=shm[index+s];
    }
    __syncthreads();
}
    
 if (threadIdx.x == 0) part_sum[blockIdx.x] = shm[0];
}

void ReduceByWarpBranch(const float* input, float* output, size_t n) {
  const int32_t thread_num_per_block = 1024;  // tuned
  const int32_t block_num = 1024;             // tuned
  float *part = NULL;
  hipMalloc((void**)&part,(1024*sizeof(float)));
  // the first pass reduce input[0:n] to part[0:block_num]
  // part_sum[i] stands for the result of i-th block
  size_t shm_size = thread_num_per_block * sizeof(float);  // float per thread
  warpBranchKernel<<<block_num, thread_num_per_block, shm_size>>>(input,part, n);
  // the second pass reduce part[0:block_num] to output
  warpBranchKernel<<<1, thread_num_per_block, shm_size>>>(part, output, block_num);
}

__global__ void TwoPassSharedOptimizedKernel(const float* input,
                                             float* part_sum, size_t n) {
  int32_t gtid = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
  int32_t total_thread_num = gridDim.x * blockDim.x;
  // reduce
  //   input[gtid + total_thread_num * 0]
  //   input[gtid + total_thread_num * 1]
  //   input[gtid + total_thread_num * 2]
  //   input[gtid + total_thread_num * ...]
  float sum = 0.0f;
  for (int32_t i = gtid; i < n; i += total_thread_num) {
    sum += input[i];
  }
  // store sum to shared memory
  extern __shared__ float shm[];
  shm[threadIdx.x] = sum;
  __syncthreads();
  // reduce shm to part_sum
  for (int32_t active_thread_num = blockDim.x / 2; active_thread_num >= 1;
       active_thread_num /= 2) {
    if (threadIdx.x < active_thread_num) {
      shm[threadIdx.x] += shm[threadIdx.x + active_thread_num];
    }
    __syncthreads();
  }
  //   Shared Memory 有 4 字节模式和 8 字节模式：
  // 4 字节模式：其中属于 Bank 0 的地址有 [0, 4), [128, 132), [256, 260)...，而属于 Bank 1 的地址有 [4, 8), [132, 136), [260, 264) ...，依次类推每个 bank 的地址。
  // 8 字节模式：其中属于 Bank 0 的地址有 [0, 8), [256, 264), [512, 520)...，而属于 Bank 1 的地址有 [8, 16), [264, 272), [520, 528) ...，依次类推每个 bank 的地址。
  //   Bank  |      1      |      2      |      3      |...
  // Address |  0  1  2  3 |  4  5  6  7 |  8  9 10 11 |...
  // Address | 64 65 66 67 | 68 69 70 71 | 72 73 74 75 |...
  //    
  // 0 + 4 , 1 + 5, 2 + 6 3 + 7
  // 0 + 2 , 1 + 3   
  if (threadIdx.x == 0) {
    part_sum[blockIdx.x] = shm[0];
  }
}

void ReduceByTwoPassSharedOptimized(const float* input, float* output,size_t n) {
  const int32_t thread_num_per_block = 1024;  // tuned
  const int32_t block_num = 1024;             // tuned
  float *part = NULL;
  hipMalloc((void**)&part,(1024*sizeof(float)));
  // the first pass reduce input[0:n] to part[0:block_num]
  // part_sum[i] stands for the result of i-th block
  size_t shm_size = thread_num_per_block * sizeof(float);  // float per thread
  TwoPassSharedOptimizedKernel<<<block_num, thread_num_per_block, shm_size>>>(input,part, n);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("ReduceByTwoPassSharedOptimized 1 CUDA Error: %s\n", hipGetErrorString(err));
  }
  // the second pass reduce part[0:block_num] to output
  TwoPassSharedOptimizedKernel<<<1, thread_num_per_block, shm_size>>>(part, output, block_num);
}


int main() 
{
    float *data = NULL;
    hipHostMalloc((void**)&data, (4*1024*1024*sizeof(float)), hipHostMallocDefault);
    for (int i=0; i<4*1024*1024; i++) {
        data[i] = (float)i;
    }

    float *input = NULL;
    float *output = NULL;
    hipMalloc((void**)&input,(4*1024*1024*sizeof(float)));
    hipMalloc((void**)&output,sizeof(float));

    // HtoD;
    hipMemcpy(input, data, 4*1024*1024*sizeof(float), hipMemcpyHostToDevice);

    ReduceBySerial(input, output, 4*1024*1024);

    ReduceByAtomic(input, output, 4*1024*1024);

    ReduceByTwoPassInterleaved(input, output, 4*1024*1024);

    ReduceByBaseline(input, output, 4*1024*1024);

    ReduceByWarpBranch(input, output, 4*1024*1024);

    ReduceByTwoPassSharedOptimized(input, output, 4*1024*1024);

    hipDeviceSynchronize();
    return 0;
}